#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <mma.h>

__device__ void int4_matmul_kernel(
    float* __restrict__ out,
    float* __restrict__ activation,
    float* __restrict__ quantized_weight,
    float* __restrict__ scales,
    float* __restrict__ zeros,
    int out_width,
    int out_height,
    int weight_width,
    int weight_height,
    int groupsize,
    int qmin,
    int qmax
){
    int matmul_dim = weight_height;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= out_height || col >= out_width) {
        return;
    }

    int mid = (int)((qmax + qmin + 1)/2);

    int sz_index = (int)(col / groupsize);
    int sz_width = (int)(weight_width / groupsize);

    for(int i = 0; i < matmul_dim; i++){
        float activation_element = activation[row * matmul_dim + i];
        float weight_element = quantized_weight[i * weight_width + col];
        weight_element-= mid;
        
        float scale_element = scales[i * sz_width + sz_index];
        weight_element *= scale_element;
        float zero_element = zeros[i * sz_width + sz_index];
        weight_element += zero_element; 

        out[row * out_width + col] += weight_element * activation_element;
    }
}

using namespace nvcuda;

__global__ void int4_matmul_unpack32_kernel(
    float* __restrict__ out,
    const __hip_bfloat16* __restrict__ activation,
    const int32_t* __restrict__ packed_weights,
    const float* __restrict__ scales,
    const float* __restrict__ zeros,
    int out_width,
    int out_height,
    int weight_width,
    int weight_height,
    int groupsize
) {
    constexpr int WMMA_M = 16;
    constexpr int WMMA_N = 8;
    constexpr int WMMA_K = 32;
    constexpr int MID = 8;  // (qmin + qmax + 1) // 2, where qmin = 0, qmax = 15

    int row = blockIdx.y * WMMA_M;
    int col = blockIdx.x * WMMA_N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __hip_bfloat16, wmma::row_major> frag_act;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, int4_t, wmma::col_major> frag_weight;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> frag_acc;

    wmma::fill_fragment(frag_acc, 0.0f);

    // Pre-determine the column in the weight matrix
    int weight_col = blockIdx.x;

    for (int k = 0; k < weight_height; k += WMMA_K) {
        // Load activation fragment (16 x 32)
        if (row < out_height) {
            wmma::load_matrix_sync(frag_act, activation + row * weight_height + k, weight_height);
        } else {
            wmma::fill_fragment(frag_act, __float2bfloat16(0.0f));
        }

        // Load and unpack weight fragment (32 x 8)
        if (k < weight_height && col < weight_width) {
            for (int i = 0; i < WMMA_K; i++) {
                // Corrected: Use k as the starting row for our 32x1 weight fragment
                int packed_idx = (i + k) * weight_width + weight_col;
                int32_t packed_val = packed_weights[packed_idx];
                
                for (int j = 0; j < WMMA_N; j++) {
                    int fragment_idx = i * WMMA_N + j;
                    if (fragment_idx < WMMA_K * WMMA_N) {
                        frag_weight.x[fragment_idx] = (packed_val >> (j * 4)) & 0xF;
                    } else {
                        frag_weight.x[fragment_idx] = 0;
                    }
                }
            }
        } else {
            wmma::fill_fragment(frag_weight, 0);
        }

        // Perform matrix multiplication (16 x 32 * 32 x 8 = 16 x 8)
        wmma::mma_sync(frag_acc, frag_act, frag_weight, frag_acc);

        for (int i = 0; i < frag_acc.num_elements; i++) {
            int out_row = row + i / WMMA_N;
            int out_col = col + i % WMMA_N;
            if (out_row < out_height && out_col < out_width) {
                int out_idx = out_row * out_width + out_col;
                int group_idx = out_col / groupsize;
                float scale = scales[out_row * (weight_width / groupsize) + group_idx];
                float zero = zeros[out_row * (weight_width / groupsize) + group_idx];
                
                // Dequantize here, accounting for the quantization scheme
                float dequantized = (frag_acc.x[i] - MID) * scale + zero;
                out[out_idx] += dequantized; //add here, don't just set. In tiling, we need to add because each 16 x 8 product is only a piece of the total.
            }
        }
    }
    // Dequantize and store results
}